#include "hip/hip_runtime.h"
// ==================== params.cu ====================
#include "params.cuh"

// Host-side variables
float h_dt;
float3 h_gravity;
float host_dx;
float host_dy;
float host_node_mass;

// Device-side constants
__constant__ float ks;
__constant__ float kd;
__constant__ float node_mass;
__constant__ float dt;
__constant__ float3 gravity;
__constant__ float cloth_width;
__constant__ float cloth_height;
__constant__ int num_x;
__constant__ int num_y;
__constant__ float dx;
__constant__ float dy;
__constant__ float plastic_threshold;

void uploadSimParamsToDevice(int Nx, int Ny, float width, float height) {
    float h_ks = 100.0f;        
    float h_kd = 2.0f;         
    float h_mass = 1.0f;
    float h_dt_val = 0.005f; // Smaller timestep for stability
    float h_dx_val = width / (Nx - 1);
    float h_dy_val = height / (Ny - 1);
    float h_threshold = 1.25f;
    float3 h_gravity_val = make_float3(0.0f, -9.81f, 0.0f);

    // Set host-side copies
    h_dt = h_dt_val;
    h_gravity = h_gravity_val;
    host_dx = h_dx_val;
    host_dy = h_dy_val;
    host_node_mass = h_mass;

    // Copy values to device constants
    hipMemcpyToSymbol(HIP_SYMBOL(ks), &h_ks, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(kd), &h_kd, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(node_mass), &h_mass, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dt), &h_dt_val, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(gravity), &h_gravity_val, sizeof(float3));
    hipMemcpyToSymbol(HIP_SYMBOL(cloth_width), &width, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(cloth_height), &height, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(num_x), &Nx, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(num_y), &Ny, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(dx), &h_dx_val, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(dy), &h_dy_val, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(plastic_threshold), &h_threshold, sizeof(float));
}
